#include <stdio.h>
#include <hip/hip_runtime.h>

__global__
void hello_world(float* a){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    a[i] = i;
}

int main(void){
    int size = 256;
    int n = size * sizeof(float);
    float *a, *g;

    a = (float*)malloc(n);
    hipMalloc((void**)&g, n);
    
    hello_world<<<1, 256>>>(g);
    hipDeviceSynchronize();
    
    hipMemcpy(a, g, n, hipMemcpyDeviceToHost);

    for(int i = 0; i < size; i++)
        printf("%f\n", a[i]);

    free(a);
    hipFree(g);

    hipDeviceReset();

    return 0;
}